#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <limits.h>

#define TOTAL_DEGREES 90
#define BINS_PER_DEG 4
#define THREADS_PER_BLK 640

// data for the real galaxies will be read into these arrays
float *ra_real, *decl_real;
// number of real galaxies
int NoofReal;

// data for the simulated random galaxies will be read into these arrays
float *ra_sim, *decl_sim;
// number of simulated random galaxies
int NoofSim;

// histograms will be stored in the following
unsigned int *histogramDR, *histogramDD, *histogramRR;

// total no. of bins in the histogram
const int NO_OF_BINS = TOTAL_DEGREES * BINS_PER_DEG;

// value of one radian
const float ONE_RAD = 180.0f / M_PI;

// Kernel to calculate the angular separation
__global__ void calculateHistogramAngles(float *ra_1, float *decl_1, float *ra_2, float *decl_2, unsigned int *hist, int N){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for partial histograms
    __shared__ unsigned int shared_hist[NO_OF_BINS];
    for (int i = threadIdx.x; i < NO_OF_BINS; i += blockDim.x) {
        shared_hist[i] = 0;
    }
    __syncthreads();
    
    // Use this condition so that shared_hist is always initialized by each thread
    // but only the valid threads (i.e. idx < N) contribute to the calculations
    if (idx < N){

        float a1 = ra_1[idx];
        float d1 = decl_1[idx];
        float a2, d2, angle;
        unsigned int bin;
        float sinfd1 = sinf(d1);
        float cosfd1 = cosf(d1);

        for (int j = 0; j < N; j++) {
            a2 = ra_2[j];
            d2 = decl_2[j];
            angle = sinfd1 * sinf(d2) + cosfd1 * cosf(d2) * cosf(a1 - a2);
            angle = fmaxf(-1.0f, fminf(angle, 1.0f)); // Clamp the angle
            angle = acosf(angle);
            bin = (int)(angle * ONE_RAD * 4.0f);

            // Update local histogram in shared memory
            atomicAdd(&shared_hist[bin % NO_OF_BINS], 1);
        }
    }
    __syncthreads();

    // Update global histogram
    for (int i = threadIdx.x; i < NO_OF_BINS; i += blockDim.x){
        atomicAdd(&hist[i], shared_hist[i]);
    }
}

int main(int argc, char *argv[])
{
    int noofblocks;
    int readdata(char *argv1, char *argv2);
    int getDevice(int deviceno);
    long int histogramDRsum = 0, histogramDDsum = 0, histogramRRsum = 0;
    double start, end, kerneltime;
    struct timeval _ttime;
    struct timezone _tzone;
    hipError_t myError;

    if (argc != 3)
    {
        printf("Usage: a.out real_data random_data\n");
        return (-1);
    }

    if (getDevice(0) != 0)
        return (-1);

    if (readdata(argv[1], argv[2]) != 0)
        return (-1);

    // allocate memory on the GPU
    // Using unified memory
    hipMallocManaged(&histogramDD, NO_OF_BINS * sizeof(unsigned int));
    hipMallocManaged(&histogramDR, NO_OF_BINS * sizeof(unsigned int));
    hipMallocManaged(&histogramRR, NO_OF_BINS * sizeof(unsigned int));

    // Initialize the memory to 0
    hipMemset(histogramDD, 0, NO_OF_BINS);
    hipMemset(histogramDR, 0, NO_OF_BINS);
    hipMemset(histogramRR, 0, NO_OF_BINS);

    noofblocks = (NoofReal + THREADS_PER_BLK - 1) / THREADS_PER_BLK;

    int sharedMemSize = NO_OF_BINS * sizeof(unsigned int);
    hipDeviceSynchronize();

    printf("# of blocks = %d , # threads in block = %d, # of total threads = %d \n", noofblocks, THREADS_PER_BLK, THREADS_PER_BLK * noofblocks);

    myError = hipGetLastError();
    if (myError != hipSuccess)
    {
        printf("ERROR: %s\n", hipGetErrorString(myError));
        exit(-1);
    }
    else
    {
        printf("No error during initialization..\n");
    }

    kerneltime = 0.0;
    gettimeofday(&_ttime, &_tzone);
    start = (double)_ttime.tv_sec + (double)_ttime.tv_usec / 1000000.;

    // copy data to the GPU -- Not needed as using unified memory

    // run the kernels on the GP
    // DD
    calculateHistogramAngles<<<noofblocks, THREADS_PER_BLK, sharedMemSize>>>(ra_real, decl_real, ra_real, decl_real, histogramDD, NoofReal);
    hipDeviceSynchronize();

    myError = hipGetLastError();
    if (myError != hipSuccess)
    {
        printf("ERROR: %s\n", hipGetErrorString(myError));
        exit(-1);
    }
    else
    {
        printf("No error during DD..\n");
    }

    
    // DR
    calculateHistogramAngles<<<noofblocks, THREADS_PER_BLK, sharedMemSize>>>(ra_real, decl_real, ra_sim, decl_sim, histogramDR, NoofReal);
    hipDeviceSynchronize();

    myError = hipGetLastError();
    if (myError != hipSuccess)
    {
        printf("ERROR: %s\n", hipGetErrorString(myError));
        exit(-1);
    }
    else
    {
        printf("No error during DR..\n");
    }

    // RR
    calculateHistogramAngles<<<noofblocks, THREADS_PER_BLK, sharedMemSize>>>(ra_sim, decl_sim, ra_sim, decl_sim, histogramRR, NoofSim);
    hipDeviceSynchronize();

    myError = hipGetLastError();
    if (myError != hipSuccess)
    {
        printf("ERROR: %s\n", hipGetErrorString(myError));
        exit(-1);
    }
    else
    {
        printf("No error during RR..\n");
    }
    
    for (int i = 0; i < NO_OF_BINS; ++i)
    {
        histogramDDsum += histogramDD[i];
    }
    printf("histogramDDsum = %ld\n", histogramDDsum);

    for (int i = 0; i < NO_OF_BINS; ++i)
    {
        histogramDRsum += histogramDR[i];
    }
    printf("histogramDRsum = %ld\n", histogramDRsum);

    for (int i = 0; i < NO_OF_BINS; ++i)
    {
        histogramRRsum += histogramRR[i];
    }
    printf("histogramRRsum = %ld\n", histogramRRsum);

    // copy the results back to the CPU -- Not needed as using unified memory

    // calculate omega values on the CPU
    float *omega = (float *)malloc(NO_OF_BINS * sizeof(float));
    memset(omega, 0.0f, NO_OF_BINS);

    for (int i = 0; i < NO_OF_BINS; i++)
    {
        if (histogramRR[i])
        {
            omega[i] = (float)(histogramDD[i] - 2.0f * histogramDR[i] + histogramRR[i]) / histogramRR[i];
        }
    }

    // Print first 5 values of the histograms
    for (int i = 0; i < 5; i++)
    {
        printf("%d -- %f: %d %d %d\n", i, omega[i], histogramDD[i], histogramDR[i], histogramRR[i]);
    }

    gettimeofday(&_ttime, &_tzone);
    end = (double)_ttime.tv_sec + (double)_ttime.tv_usec / 1000000.;
    kerneltime += end - start;
    printf("Kernel time: %lf\n", kerneltime);

    // Free Unified memory
    // Not required for unified memory
    
    return (0);
}

int readdata(char *argv1, char *argv2){
    int i, linecount;
    char inbuf[180];
    double ra, dec, phi, theta;
    FILE *infil;

    printf("   Assuming input data is given in arc minutes!\n");

    infil = fopen(argv1, "r");
    if (infil == NULL)
    {
        printf("Cannot open input file %s\n", argv1);
        return (-1);
    }

    // read the number of galaxies in the input file
    int announcednumber;
    if (fscanf(infil, "%d\n", &announcednumber) != 1)
    {
        printf(" cannot read file %s\n", argv1);
        return (-1);
    }
    linecount = 0;
    while (fgets(inbuf, 180, infil) != NULL)
        ++linecount;
    rewind(infil);

    if (linecount == announcednumber)
        printf("   %s contains %d galaxies\n", argv1, linecount);
    else
    {
        printf("   %s does not contain %d galaxies but %d\n", argv1, announcednumber, linecount);
        return (-1);
    }

    NoofReal = linecount;
    
    // Unified memory for real data
    hipMallocManaged(&ra_real, NoofReal * sizeof(float));
    hipMallocManaged(&decl_real, NoofReal * sizeof(float));

    // Initialize the memory to 0
    hipMemset(ra_real, 0, NoofReal);
    hipMemset(decl_real, 0, NoofReal);
    
    // skip the number of galaxies in the input file
    if (fgets(inbuf, 180, infil) == NULL)
        return (-1);
    i = 0;
    while (fgets(inbuf, 80, infil) != NULL)
    {
        if (sscanf(inbuf, "%lf %lf", &ra, &dec) != 2)
        {
            printf("   Cannot read line %d in %s\n", i + 1, argv1);
            fclose(infil);
            return (-1);
        }
        // Convert ra and decl from arcmin to rad
        phi = ra / 60.0 * M_PI / 180.0;
        theta = dec / 60.0 * M_PI / 180.0;

        ra_real[i] = (float)phi;
        decl_real[i] = (float)theta;
        ++i;
    }

    fclose(infil);

    if (i != NoofReal)
    {
        printf("   Cannot read %s correctly\n", argv1);
        return (-1);
    }

    infil = fopen(argv2, "r");
    if (infil == NULL)
    {
        printf("Cannot open input file %s\n", argv2);
        return (-1);
    }

    if (fscanf(infil, "%d\n", &announcednumber) != 1)
    {
        printf(" cannot read file %s\n", argv2);
        return (-1);
    }
    linecount = 0;
    while (fgets(inbuf, 80, infil) != NULL)
        ++linecount;
    rewind(infil);

    if (linecount == announcednumber)
        printf("   %s contains %d galaxies\n", argv2, linecount);
    else
    {
        printf("   %s does not contain %d galaxies but %d\n", argv2, announcednumber, linecount);
        return (-1);
    }

    NoofSim = linecount;

    // Unified memory for simulated data
    hipMallocManaged(&ra_sim, NoofSim * sizeof(float));
    hipMallocManaged(&decl_sim, NoofSim * sizeof(float));

    // Initialize the memory to 0
    hipMemset(ra_sim, 0, NoofSim);
    hipMemset(decl_sim, 0, NoofSim);

    // skip the number of galaxies in the input file
    if (fgets(inbuf, 180, infil) == NULL)
        return (-1);
    i = 0;
    while (fgets(inbuf, 80, infil) != NULL)
    {
        if (sscanf(inbuf, "%lf %lf", &ra, &dec) != 2)
        {
            printf("   Cannot read line %d in %s\n", i + 1, argv2);
            fclose(infil);
            return (-1);
        }
        // Convert ra and decl from arcmin to rad
        phi = ra / 60.0 * M_PI / 180.0;
        theta = dec / 60.0 * M_PI / 180.0;

        ra_sim[i] = (float)phi;
        decl_sim[i] = (float)theta;
        ++i;
    }

    fclose(infil);

    if (i != NoofSim)
    {
        printf("   Cannot read %s correctly\n", argv2);
        return (-1);
    }

    printf("Real data: first and last\n%f %f \n%f %f\n", 
        ra_real[0]*60.0*180.0/M_PI, decl_real[0]*60.0*180.0/M_PI,
        ra_real[NoofReal-1]*60.0*180.0/M_PI, decl_real[NoofSim-1]*60.0*180.0/M_PI);

    printf("Synth data: first and last\n%f %f \n%f %f\n", 
        ra_sim[0]*60.0*180.0/M_PI, decl_sim[0]*60.0*180.0/M_PI,
        ra_sim[NoofReal-1]*60.0*180.0/M_PI, decl_sim[NoofSim-1]*60.0*180.0/M_PI);    
    return (0);
}

int getDevice(int deviceNo)
{

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("   Found %d CUDA devices\n", deviceCount);
    if (deviceCount < 0 || deviceCount > 128)
        return (-1);
    int device;
    for (device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("      Device %s                  device %d\n", deviceProp.name, device);
        printf("         compute capability            =        %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("         totalGlobalMemory             =       %.2lf GB\n", deviceProp.totalGlobalMem / 1000000000.0);
        printf("         l2CacheSize                   =   %8d B\n", deviceProp.l2CacheSize);
        printf("         regsPerBlock                  =   %8d\n", deviceProp.regsPerBlock);
        printf("         multiProcessorCount           =   %8d\n", deviceProp.multiProcessorCount);
        printf("         maxThreadsPerMultiprocessor   =   %8d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("         sharedMemPerBlock             =   %8d B\n", (int)deviceProp.sharedMemPerBlock);
        printf("         warpSize                      =   %8d\n", deviceProp.warpSize);
        printf("         clockRate                     =   %8.2lf MHz\n", deviceProp.clockRate / 1000.0);
        printf("         maxThreadsPerBlock            =   %8d\n", deviceProp.maxThreadsPerBlock);
        printf("         asyncEngineCount              =   %8d\n", deviceProp.asyncEngineCount);
        printf("         f to lf performance ratio     =   %8d\n", deviceProp.singleToDoublePrecisionPerfRatio);
        printf("         maxGridSize                   =   %d x %d x %d\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("         maxThreadsDim in thread block =   %d x %d x %d\n",
               deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("         concurrentKernels             =   ");
        if (deviceProp.concurrentKernels == 1)
            printf("     yes\n");
        else
            printf("    no\n");
        printf("         deviceOverlap                 =   %8d\n", deviceProp.deviceOverlap);
        if (deviceProp.deviceOverlap == 1)
            printf("            Concurrently copy memory/execute kernel\n");
    }

    hipSetDevice(deviceNo);
    hipGetDevice(&device);
    if (device != 0)
        printf("   Unable to set device 0, using %d instead", device);
    else
        printf("   Using CUDA device %d\n\n", device);

    return (0);
}
